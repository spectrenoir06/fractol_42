/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   mandelbrot.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: adoussau <antoine@doussaud.org>            +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2015/02/03 16:08:27 by adoussau          #+#    #+#             */
/*   Updated: 2015/02/03 16:08:30 by adoussau         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
# define WIN_SZ_X 1024
# define WIN_SZ_Y 1024

__global__ void		mandelbrot(int	*d_tab, double offx, double offy, double zoom, int ite_max, int winszx, int winszy)
{
	double	x1;
	double	y1;
	double	c_r;
	double	c_i;
	double	z_r;
	double	z_i;
	double	d_i;
	double	d_r;
	int		i;
	int		row;
	int		col;
	int		index;
	row = blockIdx.y * blockDim.y + threadIdx.y;
	col = blockIdx.x * blockDim.x + threadIdx.x;
	index = row * winszx + col;
	if(col >= winszx || row >= winszy)
		return;

	x1 = -2.1;
	y1 = -1.2;
	c_r = (((double)col + offx) / zoom) + x1;
	c_i = (((double)row + offy) / zoom) + y1;
	z_r = c_r;
	z_i = c_i;
	d_r = z_r * z_r;
	d_i = z_i * z_i;
	i = 0;
	while((d_r + d_i) < 4 && i < ite_max)
	{
		z_i = (2 * z_r * z_i) + c_i;
		z_r = d_r - d_i + c_r;
		d_r = z_r * z_r;
		d_i = z_i * z_i;
		i++;
	}
	d_tab[index] = i;
}

extern "C" void			call_mandelbrot(int *tab, double offx, double offy, double zoom, int ite_max, int winszx, int winszy)
{
	int		*d_tab;
	int		size;
	dim3	block_size(16, 16);
	dim3	grid_size(WIN_SZ_X / block_size.x, WIN_SZ_Y / block_size.y);

	size = WIN_SZ_Y * WIN_SZ_X * sizeof(int);
	hipMalloc((void **)&d_tab, size);
	mandelbrot<<<grid_size,block_size>>>(d_tab, offx, offy, zoom, ite_max, winszx, winszy);

	hipMemcpy(tab, d_tab, size, hipMemcpyDeviceToHost);
	hipFree(d_tab);
}
