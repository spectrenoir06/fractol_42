#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   mandelbrot.cu                                      :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: adoussau <antoine@doussaud.org>            +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2015/02/03 16:08:27 by adoussau          #+#    #+#             */
/*   Updated: 2015/02/03 16:08:30 by adoussau         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
# define WIN_SZ_X 1024
# define WIN_SZ_Y 1024

# define TYPE double

__global__ void		glynn(int	*d_tab, double offx, double offy, double zoom, int ite_max, int winszx, int winszy)
{
	float	temp = 0;

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int index = row * winszx + col;
	if(col >= winszx || row >= winszy)
		return;
	TYPE z_r = (((double)col + offx) / zoom) + -2.1;
	TYPE z_i = (((double)row + offy) / zoom) + -1.2;
	unsigned int i = 0;
	while(sqrt(z_r * z_r + z_i * z_i) < 1 && i < it_max)
	{
		temp = z_r;
		z_r = sqrt((z_r * z_r - z_i * z_i) * (z_r * z_r - z_i * z_i)) + c_r;
		z_i = sqrt((2 * z_i * temp)*(2 * z_i * temp)) + c_i;
		i++;
	}
	d_tab[index] = i;
}

extern "C" void			call_glynn(int *tab, double offx, double offy, double zoom, int ite_max, int winszx, int winszy)
{
	int		*d_tab = NULL;
	int		size = 0;
	dim3	block_size(16, 16);
	dim3	grid_size(WIN_SZ_X / block_size.x, WIN_SZ_Y / block_size.y);

	size = WIN_SZ_Y * WIN_SZ_X * sizeof(int);
	hipMalloc((void **)&d_tab, size);
	glynn<<<grid_size,block_size>>>(d_tab, offx, offy, zoom, ite_max, winszx, winszy);

	hipMemcpy(tab, d_tab, size, hipMemcpyDeviceToHost);
	hipFree(d_tab);
}
