/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   julia.cu                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: adoussau <antoine@doussaud.org>            +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2015/02/03 16:08:27 by aalliot           #+#    #+#             */
/*   Updated: 2015/02/03 16:08:30 by aalliot          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
# define WIN_SZ_X 1024
# define WIN_SZ_Y 1024

__global__ void		julia(int	*d_tab, double offx, double offy, double zoom, int ite_max, int winszx, int winszy, double c_i, double c_r)
{
	double	x1;
	double	y1;
	double	z_r;
	double	z_i;
	double	tmp;
	int		i;
	int		row;
	int		col;
	int		index;
	row = blockIdx.y * blockDim.y + threadIdx.y;
	col = blockIdx.x * blockDim.x + threadIdx.x;
	index = row * winszx + col;
	if(col >= winszx || row >= winszy)
		return;

	x1 = -1.4;
	y1 = -1.2;
	z_r = (((double)col + offx) / zoom) + x1;
	z_i = (((double)row + offy) / zoom) + y1;
	i = 0;
	while((z_r * z_r + z_i * z_i) < 4 && i < ite_max)
	{
		tmp = z_r;
		z_r = (z_r * z_r) - (z_i * z_i) + c_r;
		z_i = (2 * tmp * z_i) + c_i;
		i++;
	}
	d_tab[index] = i;
}

extern "C" void		call_julia(int *tab, double offx, double offy, double zoom, int ite_max, int winszx, int winszy, double c_i, double c_r)
{
	int		*d_tab;
	int		size;
	dim3	block_size(16, 16);
	dim3	grid_size(WIN_SZ_X / block_size.x, WIN_SZ_Y / block_size.y);

	size = WIN_SZ_Y * WIN_SZ_X * sizeof(int);
	hipMalloc((void **)&d_tab, size);
	julia<<<grid_size,block_size>>>(d_tab, offx, offy, zoom, ite_max, winszx, winszy, c_i, c_r);

	hipMemcpy(tab, d_tab, size, hipMemcpyDeviceToHost);
	hipFree(d_tab);
}
